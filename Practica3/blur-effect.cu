#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

__device__ int blur_pixel(const int* image, int x, int y, int width, int height, int k){

	int ic, ir, fc, fr, n;
	x-(k/2)+1<0 ? ic = 0 : ic = x-(k/2);
	y-(k/2)+1<0 ? ir = 0 : ir = y-(k/2);
	x+(k/2)+1>width ? fc = width : fc = x+(k/2)+1;
	y+(k/2)+1>height ? fr = height : fr = y+(k/2)+1;

	int red = 0, green = 0, blue = 0;
	for(int i=ic; i<fc; i++){
    		for(int j=ir; j<fr; j++){
			n = image[j+i*height];
			blue += (n % 1000);
			green += (n/1000) % 1000;
			red += (n/1000000) % 1000;
		}
	}

	blue = blue / (k*k);
	green = green / (k*k);
	red = red / (k*k);
	return (red*1000000)+(green*1000)+blue;
}

//Funcion de cada hilo.
__global__ void blur_thread(const int* d_in, const int width, const int height, const int kernel, const int total_threads, int* d_out){
	
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int ir = id * ( height / total_threads );
	int fr = (id + 1) * ( height / total_threads );

	if(id < height){
		for(int i=0; i<width; i++){
			for(int j=ir; j<fr; j++){
				d_out[j+i*height] = blur_pixel(d_in, i, j, width, height, kernel);
			}
		}
	}
}


//Main.
int main(int argc, char** argv){

	char* src;
	Mat img, dst;
	int kernel, threads, width, height;
		
	src = argv[1];
	kernel = atoi(argv[2]);
	threads = atoi(argv[3]);
	
	if(argc != 4){
		cout<<"Numero incorrecto de argumentos.\n";
		return -1;
	}

	img = imread(src);
	if(!img.data){
		cout<<"Imagen no reconocida.\n";
		return -1;
	}

	width = img.cols;
	height = img.rows;
	dst = img.clone();
	hipError_t err = hipSuccess;

	size_t size = width * height * sizeof(int);
	int *h_in = (int *)malloc(size);
	int *h_out = (int *)malloc(size); 

	int count = 0;
	for(int i=0; i<width; i++){
		for(int j=0; j<height; j++){
			h_in[count] = img.at<Vec3b>(j,i)[0];
			h_in[count] += img.at<Vec3b>(j,i)[1] * 1000;
			h_in[count] += img.at<Vec3b>(j,i)[2] * 1000000;
			count++;
		}
	}

	int *d_in = NULL;
	err = hipMalloc((void **)&d_in, size);
	if(err != hipSuccess){
		cout<<"Error separando espacio imagen normal en GPU "<<hipGetErrorString(err)<<endl;
		return -1;
	}

	int *d_out = NULL;
	err = hipMalloc((void **)&d_out, size);
	if(err != hipSuccess){
		cout<<"Error separando espacio imagen difuminada en GPU "<<hipGetErrorString(err)<<endl;
		return -1;
	}

	err = hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);
	if (err != hipSuccess){
		cout<<"Error copiando datos a GPU "<<hipGetErrorString(err)<<endl;
		return -1;
	}

	int blocksPerGrid = (height + threads - 1) / threads;	
	blur_thread<<<blocksPerGrid, threads>>>(d_in, width, height, kernel, height, d_out);
	err = hipGetLastError();
	if (err != hipSuccess){
		cout<<"Fallo al lanzar Kerndel de GPU "<<hipGetErrorString(err)<<endl;
		return -1;
	}
	
    	err = hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		cout<<"Error copiando desde GPU a CPU "<<hipGetErrorString(err)<<endl;
		return -1;
	}

	count = 0;
	for(int i=0; i<width; i++){
		for(int j=0; j<height; j++){
			dst.at<Vec3b>(j,i)[0] = (unsigned char)((h_out[count]) % 1000);
			dst.at<Vec3b>(j,i)[1] = (unsigned char)((h_out[count]/1000) % 1000);
			dst.at<Vec3b>(j,i)[2] = (unsigned char)((h_out[count]/1000000) % 1000);
			count++;
		}
	}
	imwrite("out.jpg", dst);
	
	err = hipFree(d_in);
	if (err != hipSuccess){
	        cout<<"Error liberando memoria de imagen normal "<<hipGetErrorString(err)<<endl;
		return -1;
    	}

	err = hipFree(d_out);
	if (err != hipSuccess){
	        cout<<"Error liberando memoria de imagen difuminada "<<hipGetErrorString(err)<<endl;
		return -1;
    	}

	free(h_in);
	free(h_out);

	return 0;
}
